#include "hip/hip_runtime.h"
/*
  STREAM benchmark implementation in CUDA.

    COPY:       a(i) = b(i)                 
    SCALE:      a(i) = q*b(i)               
    SUM:        a(i) = b(i) + c(i)          
    TRIAD:      a(i) = b(i) + q*c(i)        

  It measures the memory system on the device.
  The implementation is in single precision.

  Code based on the code developed by John D. McCalpin
  http://www.cs.virginia.edu/stream/FTP/Code/stream.c

  Written by: Massimiliano Fatica, NVIDIA Corporation

  Further modifications by: Ben Cumming, CSCS
*/

#define N   (2<<26)
#define NTIMES  2

#include <stdio.h>
#include <float.h>
#include <limits.h>
#include <sys/time.h>

#include "../papi_wrap.h"

# ifndef MIN
# define MIN(x,y) ((x)<(y)?(x):(y))
# endif
# ifndef MAX
# define MAX(x,y) ((x)>(y)?(x):(y))
# endif

typedef double real;

static double   avgtime[4] = {0}, maxtime[4] = {0},
        mintime[4] = {FLT_MAX,FLT_MAX,FLT_MAX,FLT_MAX};


static char *label[4] = {"Copy:      ", "Scale:     ", "Add:       ", "Triad:     "};

static double   bytes[4] = {
    2 * sizeof(real) * N,
    2 * sizeof(real) * N,
    3 * sizeof(real) * N,
    3 * sizeof(real) * N
};

/* A gettimeofday routine to give access to the wall
   clock timer on most UNIX-like systems.  */


double mysecond()
{
    struct timeval tp;
    struct timezone tzp;
    int i = gettimeofday(&tp,&tzp);
    return ( (double) tp.tv_sec + (double) tp.tv_usec * 1.e-6 );
}


template <typename T>
__global__ void set_array(T *a,  T value, int len)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < len)
        a[idx] = value;
}

template <typename T>
__global__ void STREAM_Copy(T *a, T *b, int len)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < len)
        b[idx] = a[idx];
}

template <typename T>
__global__ void STREAM_Scale(T *a, T *b, T scale,  int len)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < len)
        b[idx] = scale* a[idx];
}

template <typename T>
__global__ void STREAM_Add( T *a, T *b, T *c,  int len)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < len)
        c[idx] = a[idx]+b[idx];
}

template <typename T>
__global__ void STREAM_Triad( T *a, T *b, T *c, T scalar, int len)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < len)
        c[idx] = a[idx]+scalar*b[idx];
}

int main()
{
    real *d_a, *d_b, *d_c;
    int j,k;
    double times[4][NTIMES];
    real scalar;

    int handle_add;

    printf(" STREAM Benchmark implementation in CUDA\n");
    printf(" Array size (%s precision) =%7.2f MB\n", sizeof(double)==sizeof(real)?"double":"single", double(N)*double(sizeof(real))/1.e6);

    /* Allocate memory on device */
    hipMalloc((void**)&d_a, sizeof(real)*N);
    hipMalloc((void**)&d_b, sizeof(real)*N);
    hipMalloc((void**)&d_c, sizeof(real)*N);

    /* Compute execution configuration */
    dim3 dimBlock(192);
    dim3 dimGrid(N/dimBlock.x );
    if( N % dimBlock.x != 0 ) dimGrid.x+=1;

    printf(" using %d threads per block, %d blocks\n",dimBlock.x,dimGrid.x);

    /* Initialize memory on the device */
    set_array<real><<<dimGrid,dimBlock>>>(d_a, 2.f, N);
    set_array<real><<<dimGrid,dimBlock>>>(d_b, .5f, N);
    set_array<real><<<dimGrid,dimBlock>>>(d_c, .5f, N);


    handle_add =  pw_new_collector("dummy");
    pw_start_collector(handle_add);
    pw_stop_collector(handle_add);

    handle_add =  pw_new_collector("add");

    /*  --- MAIN LOOP --- repeat test cases NTIMES times --- */

    scalar=3.0f;
    for (k=0; k<NTIMES; k++)
    {
        times[0][k]= mysecond();
        STREAM_Copy<real><<<dimGrid,dimBlock>>>(d_a, d_c, N);
        hipDeviceSynchronize();
        times[0][k]= mysecond() -  times[0][k];

        times[1][k]= mysecond();
        STREAM_Scale<real><<<dimGrid,dimBlock>>>(d_b, d_c, scalar,  N);
        hipDeviceSynchronize();
        times[1][k]= mysecond() -  times[1][k];

        if(k==1) pw_start_collector(handle_add);

        times[2][k]= mysecond();
        STREAM_Add<real><<<dimGrid,dimBlock>>>(d_a, d_b, d_c,  N);
        hipDeviceSynchronize();
        times[2][k]= mysecond() -  times[2][k];

        if(k==1) pw_stop_collector(handle_add);

        times[3][k]= mysecond();
        STREAM_Triad<real><<<dimGrid,dimBlock>>>(d_b, d_c, d_a, scalar,  N);
        hipDeviceSynchronize();
        times[3][k]= mysecond() -  times[3][k];
    }

    pw_print();
    /*  --- SUMMARY --- */

    for (k=1; k<NTIMES; k++) /* note -- skip first iteration */
    {
        for (j=0; j<4; j++)
        {
            avgtime[j] = avgtime[j] + times[j][k];
            mintime[j] = MIN(mintime[j], times[j][k]);
            maxtime[j] = MAX(maxtime[j], times[j][k]);
        }
    }

    printf("Function      Rate (GB/s)   Avg time(s)  Min time(s)  Max time(s)\n");
    for (j=0; j<4; j++) {
        avgtime[j] = avgtime[j]/(double)(NTIMES-1);

        printf("%s%11.4f     %11.8f  %11.8f  %11.8f\n", label[j],
                1.0E-09 * bytes[j]/mintime[j],
                avgtime[j],
                mintime[j],
                maxtime[j]);
    }


    /* Free memory on device */
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

}
